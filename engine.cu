#include "hip/hip_runtime.h"
#include <assert.h>
#include <math.h>
#include <stdio.h>
#include <iostream>
#include <random>

#define MAX_DENSITY (64)

#define MEAN_VELOCITY (0.02)
#define DAMPING (0.999)
#define DT (0.01)

typedef struct Particle
{
    float pos[DIMS];
    float vel[DIMS];
} Particle;

typedef struct Cell
{
    int count;
    Particle particles[MAX_DENSITY];
} Cell;


__device__
void collide(Particle *a, Particle *b) {
    // DIMS = 2
    float dx = a->pos[0] - b->pos[0];
    float dy = a->pos[1] - b->pos[1];

    float distance = sqrt(dx*dx + dy*dy);
    if (distance < RADIUS * GRID_WIDTH) {
        dx /= distance;
        dy /= distance;

        float vx = a->vel[0] - b->vel[0];
        float vy = a->vel[1] - b->vel[1];

        float u = (dx * vx + dy * vy) * DAMPING;
        float ux = dx * u;
        float uy = dy * u;

        a->vel[0] -= ux;
        a->vel[1] -= uy;
        b->vel[0] += ux;
        b->vel[1] += uy;

        float excess = RADIUS * GRID_WIDTH - distance;

        a->pos[0] += 0.5 * excess * dx;
        a->pos[1] += 0.5 * excess * dy;
        b->pos[0] -= 0.5 * excess * dx;
        b->pos[1] -= 0.5 * excess * dy;
    }
}

__global__
void step(Cell *cells) {
    // DIMS = 2
    int index = threadIdx.x + GRID_WIDTH * blockIdx.x;

    // Move particles
    for (int i = 0; i < cells[index].count; ++i) {
        for (int j = 0; j < DIMS; ++j) {
            cells[index].particles[i].pos[j] += DT * cells[index].particles[i].vel[j];
        }
    }

    // Figure out grid neighbours
    // Diagonals intentionally omitted in favor of efficiency at the cost of accuracy
    int west = index - 1;
    if (threadIdx.x == 0) {
        west = GRID_WIDTH - 1 + GRID_WIDTH * blockIdx.x;
    }
    int north = index - GRID_WIDTH;
    if (blockIdx.x == 0) {
        north = threadIdx.x + GRID_WIDTH * (GRID_WIDTH - 1);
    }

    // We only need pairwise collisions so we can omit the reverse pass over the grid.
    // int east = index + 1;
    // if (threadIdx.x == GRID_WIDTH - 1) {
    //     east = blockIdx.x * GRID_WIDTH;
    // }
    // int south = index + GRID_WIDTH;
    // if (blockIdx.x == GRID_WIDTH - 1) {
    //     south = threadIdx.x;
    // }

    for (int i = 0; i < cells[index].count; ++i) {
        for (int j = i + 1; j < cells[index].count; ++j) {
            collide(cells[index].particles + i, cells[index].particles + j);
        }
        for (int j = 0; j < cells[west].count; ++j) {
            collide(cells[index].particles + i, cells[west].particles + j);
        }
        for (int j = 0; j < cells[north].count; ++j) {
            collide(cells[index].particles + i, cells[north].particles + j);
        }
        // See comment about pairwise collisions above on why these can be omitted.
        // for (int j = 0; j < cells[east].count; ++j) {
        //     collide(cells[index].particles + i, cells[east].particles + j);
        // }
        // for (int j = 0; j < cells[south].count; ++j) {
        //     collide(cells[index].particles + i, cells[south].particles + j);
        // }
    }
}

void migrate(Cell *cells) {
    for (int i = 0; i < GRID_WIDTH; ++i) {
        for (int j = 0; j < GRID_WIDTH; ++j) {
            int index = i + GRID_WIDTH * j;
            int west = i - 1 + GRID_WIDTH * j;
            if (i == 0) {
                west = GRID_WIDTH - 1 + GRID_WIDTH * j;
            }
            int east = i + 1 + GRID_WIDTH * j;
            if (i == GRID_WIDTH - 1) {
                east = GRID_WIDTH * j;
            }
            int north = i + (j - 1) * GRID_WIDTH;
            if (j == 0) {
                north = i + GRID_WIDTH * (GRID_WIDTH - 1);
            }
            int south = i + (j + 1) * GRID_WIDTH;
            if (j == GRID_WIDTH - 1) {
                south = i;
            }

            // There are no diagonal migrations. It is hoped that those will resolve themselves in two steps.
            // assert(cells[index].count >= 0);
            // assert(cells[index].count <= MAX_DENSITY);
            for (int k = 0; k < cells[index].count; ++k) {
                int drop = 0;
                if (cells[index].particles[k].pos[0] < i) {
                    if (cells[west].count == MAX_DENSITY) {
                        continue;
                    }
                    if (cells[index].particles[k].pos[0] < 0) {
                        cells[index].particles[k].pos[0] += GRID_WIDTH;
                    }
                    cells[west].particles[cells[west].count] = cells[index].particles[k];
                    cells[west].count += 1;
                    drop = 1;
                } else if (cells[index].particles[k].pos[0] >= i + 1) {
                    if (cells[east].count == MAX_DENSITY) {
                        continue;
                    }
                    if (cells[index].particles[k].pos[0] >= GRID_WIDTH) {
                        cells[index].particles[k].pos[0] -= GRID_WIDTH;
                    }
                    cells[east].particles[cells[east].count] = cells[index].particles[k];
                    cells[east].count += 1;
                    drop = 1;
                } else if (cells[index].particles[k].pos[1] < j) {
                    if (cells[north].count == MAX_DENSITY) {
                        continue;
                    }
                    if (cells[index].particles[k].pos[1] < 0) {
                        cells[index].particles[k].pos[1] += GRID_WIDTH;
                    }
                    cells[north].particles[cells[north].count] = cells[index].particles[k];
                    cells[north].count += 1;
                    drop = 1;
                } else if (cells[index].particles[k].pos[1] >= j + 1) {
                    if (cells[south].count == MAX_DENSITY) {
                        continue;
                    }
                    if (cells[index].particles[k].pos[1] >= GRID_WIDTH) {
                        cells[index].particles[k].pos[1] -= GRID_WIDTH;
                    }
                    cells[south].particles[cells[south].count] = cells[index].particles[k];
                    cells[south].count += 1;
                    drop = 1;
                }

                if (drop) {
                    cells[index].count -= 1;
                    for (int l = k; l < cells[index].count; ++l) {
                        cells[index].particles[l] = cells[index].particles[l + 1];
                    }
                }
            }
        }
    }
}

void manipulate(Cell *cells)
{
    for (int index = 0; index < GRID_WIDTH * GRID_WIDTH; ++index) {
        for (int k = 0; k < cells[index].count; ++k) {
            Particle p = cells[index].particles[k];
            if (p.pos[0] < GRID_WIDTH * 0.2 && p.pos[1] > GRID_WIDTH * 0.45 && p.pos[1] < GRID_WIDTH * 0.55) {
                cells[index].particles[k].vel[0] += DT * GRID_WIDTH * 0.03;
            }

            if (p.pos[0] < GRID_WIDTH * 0.01 && p.vel[0] < 0) {
                cells[index].particles[k].vel[0] = -p.vel[0];
            }
            if (p.pos[0] > GRID_WIDTH - GRID_WIDTH * 0.01 && p.vel[0] > 0) {
                cells[index].particles[k].vel[0] = -p.vel[0];
            }
            if (p.pos[1] < GRID_WIDTH * 0.01 && p.vel[1] < 0) {
                cells[index].particles[k].vel[1] = -p.vel[1];
            }
            if (p.pos[1] > GRID_WIDTH - GRID_WIDTH * 0.01 && p.vel[1] > 0) {
                cells[index].particles[k].vel[1] = -p.vel[1];
            }
        }
    }
}

int main(void)
{
    Cell *cells;

    std::random_device dev;
    std::mt19937 rng(dev());
    std::normal_distribution<float> ndist(0, 1);
    std::uniform_real_distribution<float> udist(0, 1);

    // DIMS = 2
    hipMallocManaged(&cells, GRID_WIDTH * GRID_WIDTH);

    for (int i = 0; i < GRID_WIDTH; ++i) {
        for (int j = 0; j < GRID_WIDTH; ++j) {
            int index = i + GRID_WIDTH * j;
            cells[index].count = DENSITY;
            for (int k = 0; k < cells[index].count; ++k) {
                cells[index].particles[k].pos[0] = i + udist(rng);
                cells[index].particles[k].pos[1] = j + udist(rng);
                cells[index].particles[k].vel[0] = ndist(rng) * MEAN_VELOCITY * GRID_WIDTH;
                cells[index].particles[k].vel[1] = ndist(rng) * MEAN_VELOCITY * GRID_WIDTH;
            }
        }
    }


    for (int j = 0; j < ROUNDS; ++j) {
        hipDeviceSynchronize();
        step<<<GRID_WIDTH, GRID_WIDTH>>>(cells);
        hipDeviceSynchronize();
        manipulate(cells);
        migrate(cells);

        int total_count = 0;
        for (int index = 0; index < GRID_WIDTH * GRID_WIDTH; ++index) {
            total_count += cells[index].count;
            for (int k = 0; k < cells[index].count; ++k) {
                fwrite(cells[index].particles[k].pos, sizeof(float), DIMS, stdout);
            }
        }
        assert(total_count == GRID_WIDTH*GRID_WIDTH * DENSITY);
    }

    hipFree(cells);

    return 0;
}
